/*%****************************************************************************80
%  Code: 
%   mm.cu
%
%  Purpose:
%   Parallel code matrix multiply in C 
%   with CUDA
%
%  Modified:
%   Jan 16 2022 13:15 
%
%  Author:
%    Murilo Do Carmo Boratto [murilo.boratto 'at' fieb.org.br]
%
%  How to Compile:
%    nvcc mm.c -o object
%
%  How to Execute: 
%    ./mm <size> <blocksize>
%    ./mm   16       2
%    
%****************************************************************************80*/

#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void kernel(int *A, int *B, int *C, int n) {
  
int i = blockIdx.x * blockDim.x + threadIdx.x;
int j = blockIdx.y * blockDim.y + threadIdx.y;

if(i < n && j < n)
    for( int k = 0; k < n; k++) 
       C[i*n+j]+=A[i*n+k]*B[k*n+j];

}
 

void mult_matrix_cpu(int *A, int *B, int *C, int n) {

 for(int i = 0; i < n; i++) 
     for(int j = 0; j < n; j++)
        for(int k = 0; k < n; k++) 
           C[i*n+j]+=A[i*n+k]*B[k*n+j];
        
}

void initialize_matrix(int *A, int n){
 
  for(int i=0; i < n; i++)
    for(int j=0; j < n; j++)
       A[i*n+j] = rand()%(10-1)*1;
   
}


void print_matrix(int *A, int n){

  for(int i = 0; i < n; i++){
    for(int j = 0; j < n; j++)
      printf("%d\t", A[i*n+j]);
    printf("\n");
  }

 printf("\n");

}


int main(int argc, char **argv){

    int n = atoi(argv[1]);
    int blocksize = atoi(argv[2]);

    int *A_host=(int *) malloc (n*n*sizeof(int));
    int *B_host=(int *) malloc (n*n*sizeof(int));
    int *C_host=(int *) malloc (n*n*sizeof(int));
        
    initialize_matrix(A_host,n);
    initialize_matrix(B_host,n);
      
    print_matrix(A_host,n);
    print_matrix(B_host,n);

    int *A_device;
    int *B_device;
    int *C_device;
	
    hipMalloc( (void**)&A_device, n*n*sizeof(int) ); 
    hipMalloc( (void**)&B_device, n*n*sizeof(int) ); 
    hipMalloc( (void**)&C_device, n*n*sizeof(int) ); 
	
    hipMemcpy( A_device, A_host, n*n*sizeof(int), hipMemcpyHostToDevice ); 
    hipMemcpy( B_device, B_host, n*n*sizeof(int), hipMemcpyHostToDevice ); 
	
    /*(GRID: 2D BLOCK: 2D) */
    dim3 dimGrid ( (int) ceil( (float) n / blocksize), (int) ceil( (float)n / blocksize) );
    dim3 dimBlock( blocksize, blocksize);  

    kernel<<< dimGrid,dimBlock >>>(A_device, B_device, C_device, n);        

    hipMemcpy( C_host, C_device, n*n*sizeof(int), hipMemcpyDeviceToHost ); 

    print_matrix( C_host, n );

    hipFree( A_device );
    hipFree( B_device );
    hipFree( C_device );

    free( A_host );
    free( B_host );
    free( C_host );
    
    return 0;

}

